#include <THW/THCTensor.h>

#include <nnutils/gpu/adaptive_avgpool_2d.h>
#include <torch/adaptive_avgpool_2d.h>

extern "C" {
#include <pytorch/src/gpu/adaptive_avgpool_2d.h>
}

using nnutils::THW::ConstTensor;
using nnutils::THW::MutableTensor;
using nnutils::THW::TensorTraits;

extern THCState* state;  // Defined by PyTorch

#include <iostream>

namespace nnutils {
namespace pytorch {
namespace gpu {

template <typename T>
class AdaptiveAvgpool2dCaller : public torch::AdaptiveAvgpool2dCaller<T> {
 public:
  void Forward(
      const long N, const long C, const long H, const long W,
      const long Hout, const long Wout, const long* sizes, const T* input,
      T* output) const override {
    hipStream_t stream = THCState_getCurrentStream(state);
    ::nnutils::gpu::adaptive_avgpool_2d_fwd(
         N, C, H, W, sizes, Hout, Wout, input, output, stream);
  }

  void Backward(
      const long N, const long C, const long H, const long W,
      const long Hout, const long Wout, const long* sizes, const T* g_output,
      T* g_input) const override {
    hipStream_t stream = THCState_getCurrentStream(state);
    ::nnutils::gpu::adaptive_avgpool_2d_bwd(
         N, C, H, W, sizes, Hout, Wout, g_output, g_input, stream);
  }
};

}  // namespace gpu
}  // namespace pytorch
}  // namespace nnutils


#define DEFINE_WRAPPER(TSNAME, TTYPE, TITYPE)                           \
  void nnutils_adaptive_avgpool_2d_fwd_gpu_##TSNAME(                    \
      const TTYPE* input, const TITYPE* sizes,                          \
      long int h, long int w, TTYPE* output) {                          \
    typedef typename TensorTraits<TTYPE>::DType DType;                  \
    ConstTensor<TITYPE> msizes(sizes, state);                           \
    ConstTensor<TTYPE> minput(input, state);                            \
    MutableTensor<TTYPE> moutput(output, state);                        \
    ::nnutils::torch::adaptive_avgpool_2d_fwd<TTYPE, TITYPE>(           \
         h, w, msizes, minput, &moutput,                                \
         ::nnutils::pytorch::gpu::AdaptiveAvgpool2dCaller<DType>());    \
  }                                                                     \
                                                                        \
  void nnutils_adaptive_avgpool_2d_bwd_gpu_##TSNAME(                    \
      const TTYPE* grad_output, const TITYPE* sizes,                    \
      TTYPE* grad_input) {                                              \
    typedef typename TensorTraits<TTYPE>::DType DType;                  \
    ConstTensor<TITYPE> msizes(sizes, state);                           \
    ConstTensor<TTYPE> mgrad_output(grad_output, state);                \
    MutableTensor<TTYPE> mgrad_input(grad_input, state);                \
    ::nnutils::torch::adaptive_avgpool_2d_bwd<TTYPE, TITYPE>(           \
         msizes, mgrad_output, &mgrad_input,                            \
         ::nnutils::pytorch::gpu::AdaptiveAvgpool2dCaller<DType>());    \
  }

DEFINE_WRAPPER(f32, THCudaTensor, THCudaLongTensor)
DEFINE_WRAPPER(f64, THCudaDoubleTensor, THCudaLongTensor)
